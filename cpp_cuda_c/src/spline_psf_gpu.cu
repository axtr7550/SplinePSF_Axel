#include "hip/hip_runtime.h"
//
//  Created by Lucas Müller on 12.02.2020
//  Copyright © 2020 Lucas-Raphael Müller. All rights reserved.
//
#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdexcept>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "spline_psf_gpu.cuh"
using namespace spline_psf_gpu;


// internal declarations
void check_host_coeff(const float *h_coeff);

auto forward_rois(spline *d_sp, float *d_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const bool normalize) -> void;

auto forward_drv_rois(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const float *d_bg, const bool add_bg) -> void;

/*
template <unsigned int blockSize>
__global__
void reduce_dot_product_matrix(float* g_idata_a, float* g_idata_b, float* g_odata,
unsigned int stride, unsigned int start, unsigned int n);
*/
template <int blockSize>

__device__ 
void warpReduce(volatile float* sdata, int tid) ;

__global__ 
void reduce_small(float *rois, float *g_odata, int r, const int np, int n_blocks, const int startPos);

__global__ 
void normalize(float *rois, float factor, int r, const int n_pixels);

__global__ 
void  get_factor(const float* phot_, float *total_sum, float *factors);

__device__
auto kernel_computeDelta3D(spline *sp,
    float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf,
    float x_delta, float y_delta, float z_delta) -> void;

__global__
auto kernel_derivative(spline *sp, float *rois, float *drv_rois, const int roi_ix, const int npx,
    const int npy, int xc, int yc, int zc, const float phot, const float bg,
    const float x_delta, const float y_delta, const float z_delta, const bool add_bg) -> void;

__global__
auto fAt3Dj(spline *sp, float* rois, int roi_ix, int npx, int npy,
    int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta) -> void;

__global__
auto kernel_roi(spline *sp, float *rois, const int npx, const int npy,
    const float* xc_, const float* yc_, const float* zc_, const float* phot_) -> void;

__global__
void kernel_sum_up(spline *sp, float *rois, float *sum_array, const int npx, const int npy);

__global__ 
void kernel_normalize(float *rois, float *factors, int npx, int npy);

__global__
auto kernel_derivative_roi(spline *sp, float *rois, float *drv_rois, const int npx, const int npy,
    const float *xc_, const float *yc_, const float *zc_,
    const float *phot_, const float *bg_, const bool add_bg) -> void;

__global__
auto roi_accumulate(float *frames, const int frame_size_x, const int frame_size_y, const int n_frames,
    const float *rois, const int n_rois,
    const int *frame_ix, const int *x0, const int *y0,
    const int roi_size_x, const int roi_size_y) -> void;



namespace spline_psf_gpu {

    // check cuda availability by device count
    auto cuda_is_available(void) -> bool {

        int d_count = 0;
        hipError_t err = hipGetDeviceCount(&d_count);

        if (err != hipSuccess) {
            return false;
        }

        float min_compute_cap = 3.7;

        bool at_least_one_device = false;
        for (int i = 0; i < d_count; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);

            float compute_cap = prop.major + prop.minor / 10;

            if (compute_cap >= min_compute_cap) {
                at_least_one_device = true;
                break;
            }

        }

        if (at_least_one_device) {
            return true;
        }

        return false;
    }

    // Create struct and ship it to device
    auto d_spline_init(const float *h_coeff, int xsize, int ysize, int zsize, int device_ix) -> spline* {

        // allocate struct on host and ship it to device later
        // ToDo: C++11ify this
        spline* sp;
        sp = (spline *)malloc(sizeof(spline));

        sp->xsize = xsize;
        sp->ysize = ysize;
        sp->zsize = zsize;

        sp->roi_out_eps = 1e-10;
        sp->roi_out_deriv_eps = 0.0;

        sp->n_par = 5;
        sp->n_coeff = 64;

        int tsize = xsize * ysize * zsize * 64;

        hipSetDevice(device_ix);

        float *d_coeff;
        hipMalloc(&d_coeff, tsize * sizeof(float));
        hipMemcpy(d_coeff, h_coeff, tsize * sizeof(float), hipMemcpyHostToDevice);

        sp->coeff = d_coeff;  // for some reason this should happen here and not d_sp->coeff = d_coeff ...

        // ship to device
        spline* d_sp;
        hipMalloc(&d_sp, sizeof(spline));
        hipMemcpy(d_sp, sp, sizeof(spline), hipMemcpyHostToDevice);

        // delete on host
        free(sp);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during allocation of spline struct on device.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        return d_sp;
    }

    auto destructor(spline *d_sp) -> void {

        // first create host helper to be able to access the pointer to coeff, dereferencing d_sp is illegal
        spline *sp;
        sp = (spline*)malloc(sizeof(spline));
        hipMemcpy(sp, d_sp, sizeof(spline), hipMemcpyDeviceToHost);

        hipFree(sp->coeff);
        free(sp);
        hipFree(d_sp);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during destruxtor.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }
    }


    // Wrapper function to compute the ROIs on the device.
    // Takes in all the host arguments and returns leaves the ROIs on the device
    //
    auto forward_rois_host2device(spline *d_sp, const int n, const int roi_size_x, const int roi_size_y,
    const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const bool normalize) -> float* {

        hipError_t err;

        // allocate and copy coordinates and photons
        float *d_x, *d_y, *d_z, *d_phot;
        hipMalloc(&d_x, n * sizeof(float));
        hipMalloc(&d_y, n * sizeof(float));
        hipMalloc(&d_z, n * sizeof(float));
        hipMalloc(&d_phot, n * sizeof(float));
        hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_z, h_z, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_phot, h_phot, n * sizeof(float), hipMemcpyHostToDevice);

        // allocate space for rois on device
        float* d_rois;
        hipMalloc(&d_rois, n * roi_size_x * roi_size_y * sizeof(float));

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipMemset(d_rois, 0.0, n * roi_size_x * roi_size_y * sizeof(float));

        #if DEBUG
            check_spline<<<1,1>>>(d_sp);
            hipDeviceSynchronize();
        #endif

        // call to actual implementation
        forward_rois(d_sp, d_rois, n, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot, normalize);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        hipFree(d_phot);

        return d_rois;
    }

    // Wrapper function to ocmpute the ROIs on the device and ships it back to the host
    // Takes in all the host arguments and returns the ROIs to the host
    // Allocation for rois must have happened outside
    //
    auto forward_rois_host2host(spline *d_sp, float *h_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const bool normalize) -> void {

        auto d_rois = forward_rois_host2device(d_sp, n, roi_size_x, roi_size_y, h_x, h_y, h_z, h_phot, normalize);

        hipMemcpy(h_rois, d_rois, n * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_rois);
        return;
    }

    auto forward_drv_rois_host2device(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const float *h_bg, const bool add_bg) -> void {

        // allocate and copy coordinates and photons
        float *d_x, *d_y, *d_z, *d_phot, *d_bg;
        hipMalloc(&d_x, n * sizeof(float));
        hipMalloc(&d_y, n * sizeof(float));
        hipMalloc(&d_z, n * sizeof(float));
        hipMalloc(&d_phot, n * sizeof(float));
        hipMalloc(&d_bg, n * sizeof(float));

        hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_z, h_z, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_phot, h_phot, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_bg, h_bg, n * sizeof(float), hipMemcpyHostToDevice);

        const int n_par = 5;
        hipMemset(d_rois, 0.0, n * roi_size_x * roi_size_y * sizeof(float));
        hipMemset(d_drv_rois, 0.0, n_par * n * roi_size_x * roi_size_y * sizeof(float));

        // call to actual implementation
        forward_drv_rois(d_sp, d_rois, d_drv_rois, n, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot, d_bg, add_bg);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        hipFree(d_phot);
        hipFree(d_bg);

        return;
    }

    auto forward_drv_rois_host2host(spline *d_sp, float *h_rois, float *h_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const float *h_bg, const bool add_bg) -> void {

        hipError_t err;

        // allocate space for rois and derivatives on device
        const int n_par = 5;
        float *d_rois, *d_drv_rois;

        hipMalloc(&d_rois, n * roi_size_x * roi_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipMalloc(&d_drv_rois, n_par * n * roi_size_x * roi_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during derivative ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        // forward
        forward_drv_rois_host2device(d_sp, d_rois, d_drv_rois, n, roi_size_x, roi_size_y, h_x, h_y, h_z, h_phot, h_bg, add_bg);

        hipMemcpy(h_rois, d_rois, n * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_drv_rois, d_drv_rois, n * n_par * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_rois);
        hipFree(d_drv_rois);

        return;
    }

    auto forward_frames_host2host(spline *d_sp, float *h_frames, const int frame_size_x, const int frame_size_y, const int n_frames,
        const int n_rois, const int roi_size_x, const int roi_size_y,
        const int *h_frame_ix, const float *h_xr0, const float *h_yr0, const float *h_z0,
        const int *h_x_ix, const int *h_y_ix, const float *h_phot, const bool normalize) -> void {

        auto d_frames = forward_frames_host2device(d_sp, frame_size_x, frame_size_y, n_frames,
            n_rois, roi_size_x, roi_size_y, h_frame_ix, h_xr0, h_yr0, h_z0, h_x_ix, h_y_ix, h_phot, normalize);

        hipMemcpy(h_frames, d_frames, n_frames * frame_size_x * frame_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_frames);
        return;
    }

    auto forward_frames_host2device(spline *d_sp, const int frame_size_x, const int frame_size_y, const int n_frames,
        const int n_rois, const int roi_size_x, const int roi_size_y,
        const int *h_frame_ix, const float *h_xr0, const float *h_yr0, const float *h_z0,
        const int *h_x_ix, const int *h_y_ix, const float *h_phot, const bool normalize) -> float* {

        hipError_t err;

        // ToDo: maybe convert to stream
        float* d_frames;

        // Creates d_frames on GPU to store all the frames

        hipMalloc(&d_frames, n_frames * frame_size_x * frame_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during Frame memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }
        hipMemset(d_frames, 0.0, n_frames * frame_size_x * frame_size_y * sizeof(float));

        // allocate indices
        int *d_xix, *d_yix, *d_fix;
        hipMalloc(&d_xix, n_rois * sizeof(int));
        hipMalloc(&d_yix, n_rois * sizeof(int));
        hipMalloc(&d_fix, n_rois * sizeof(int));
        hipMemcpy(d_xix, h_x_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_yix, h_y_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_fix, h_frame_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);

        auto d_rois = forward_rois_host2device(d_sp, n_rois, roi_size_x, roi_size_y, h_xr0, h_yr0, h_z0, h_phot, normalize);

        // accumulate rois into frames
        const int blocks = (n_rois * roi_size_x * roi_size_y) / 256 + 1;
        const int thread_p_block = 256;
        // One thread per roi pixel
        // Still quite quick
        roi_accumulate<<<blocks, thread_p_block>>>(d_frames, frame_size_x, frame_size_y, n_frames,
        d_rois, n_rois, d_fix, d_xix, d_yix, roi_size_x, roi_size_y);

        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during frame computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipFree(d_xix);
        hipFree(d_yix);
        hipFree(d_fix);
        hipFree(d_rois);

        return d_frames;
    }
} // namespace spline_psf_gpu


auto forward_rois(spline *d_sp, float *d_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const bool normalize) -> void {
    
    // printf("IN CUDA! \n");

    // init cuda_err
    hipError_t err = hipSuccess;


    // start n blocks which itself start threads corresponding to the number of px childs (dynamic parallelism)
    kernel_roi<<<n, 1>>>(d_sp, d_rois, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot);
    hipDeviceSynchronize();


    
    if (normalize){

    float *sum_array;

    //int stride = ((roi_size_x*roi_size_y)/1024);

    hipMalloc(&sum_array, n*sizeof(float));
    hipMemset(sum_array, 0.0, n*sizeof(float));



    kernel_sum_up<<<n, 1>>>(d_sp, d_rois, sum_array, roi_size_x, roi_size_y);
    hipDeviceSynchronize();


    // A bit of cheating, will work when n-elements is below 2048

    float *factors_array;

    hipMalloc(&factors_array, n*sizeof(float));
    hipMemset(factors_array, 0.0, n*sizeof(float));

    get_factor<<<n, 1>>>(d_phot, sum_array, factors_array);
    hipDeviceSynchronize();

    kernel_normalize<<<n, 1>>>(d_rois, factors_array, roi_size_x, roi_size_y);
    
    hipFree(sum_array);

    hipFree(factors_array);
    }
    // Sum array should contain sum from each block. Sum each roi and get factor. Then in parallel multiply each pixel by factor.
    

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::stringstream rt_err;
        rt_err << "Error during ROI computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
        throw std::runtime_error(rt_err.str());
    }

    return;
}

auto forward_drv_rois(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const float *d_bg, const bool add_bg) -> void {

    // init cuda_err
    hipError_t err = hipSuccess;

    // start n blocks which itself start threads corresponding to the number of px childs (dynamic parallelism)
    kernel_derivative_roi<<<n, 1>>>(d_sp, d_rois, d_drv_rois, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot, d_bg, add_bg);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::stringstream rt_err;
        rt_err << "Error during ROI derivative computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
        throw std::runtime_error(rt_err.str());
    }

    return;
}

// Just a dummy for checking correct parsing from python
// ... had to learn the hard way ...
__global__
auto check_spline(spline *d_sp) -> void {
    printf("Checking spline ...\n");
    printf("\txs, ys, zs: %i %i %i\n", d_sp->xsize, d_sp->ysize, d_sp->zsize);
    printf("\toutside-roi value: %f\n", d_sp->roi_out_eps);
    printf("\toutside-roi derivative value: %f\n", d_sp->roi_out_deriv_eps);

    printf("\tDevice coeff: \n");
    for (int i = 0; i < 100; i++) {
        printf("\t\ti: %d coeff %f\n", d_sp->coeff[i]);
    }
    printf("\n");
}

// kernel to compute common term for spline function (for all pixels this will stay the same)
__device__
auto kernel_computeDelta3D(spline *sp, float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf,
    float x_delta, float y_delta, float z_delta) -> void {
    // Should be parallizable
    int i,j,k;
    float cx,cy,cz;
    // Computes the deltas that will be multiplied by the "a" parameters delta_f is 64 long
    cz = 1.0;
    for(i=0;i<4;i++){
        cy = 1.0;
        for(j=0;j<4;j++){
            cx = 1.0;
            for(k=0;k<4;k++){
                delta_f[i*16+j*4+k] = cz * cy * cx;
                if(k<3){
					delta_dxf[i*16+j*4+k+1] = ((float)k+1) * cz * cy * cx;
				}
				if(j<3){
					delta_dyf[i*16+(j+1)*4+k] = ((float)j+1) * cz * cy * cx;
				}
				if(i<3){
					delta_dzf[(i+1)*16+j*4+k] = ((float)i+1) * cz * cy * cx;
				}
                cx = cx * x_delta;
            }
            cy = cy * y_delta;
        }
        cz= cz * z_delta;
    }
}

// kernel to compute pixel-wise term
__global__
auto fAt3Dj(spline *sp, float* rois, const int roi_ix, const int npx, const int npy,
    int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta) -> void {
    
    // The kernel knows its index and the blockdim, i.e. number of threads per block (???)
    const int i = (blockIdx.x * blockDim.x + threadIdx.x) / npx;
    const int j = (blockIdx.x * blockDim.x + threadIdx.x) % npx;

     // allocate space for df, dxf, dyf, dzf
    __shared__ float delta_f[64];
    __shared__ float dxf[64];
    __shared__ float dyf[64];
    __shared__ float dzf[64];

    // term common to all pixels, must be executed at least once per kernel block (since sync only syncs within block)
    // if (i == 0 and j == 0) {  // linear / C++ equivalent
    
    if (threadIdx.x == 0) {
        
        // 64 -- number of "a" parameters I guess this is a way of initializing the array
        for (int k = 0; k < 64; k++) {
            delta_f[k] = 0.0;
            dxf[k] = 0.0;
            dyf[k] = 0.0;
            dzf[k] = 0.0;
        }
        

        // This is different to the C library since we needed to rearrange a bit to account for the GPU parallelism
        // Could possibly be done in parallel but does not seem faster
        kernel_computeDelta3D(sp, delta_f, dxf, dyf, dzf, x_delta, y_delta, z_delta);
    }

    __syncthreads();  // wait so that all threads see the deltas. REMINDER: only works for within block 

    // kill excess threads (I think it needs to happen after syncthreads)
    if ((i >= npx) || (j >= npy)) {
        return;
    }

    // xc must originally be in like the top left corner? Unless i and j can be negative

    xc = xc + i;
    yc = yc + j;

    // If the lateral position is outside the calibration, return epsilon value
    if ((xc < 0) || (xc > sp->xsize-1) || (yc < 0) || (yc > sp->ysize-1)) {

        rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps;
        return;
    }

    zc = max(zc,0);
    zc = min(zc,sp->zsize-1);

    float fv = 0.0;
    // Loop but variables in shared memory means still fast
    for (int k = 0; k < 64; k++) {
        fv += delta_f[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
    }

    // This looks like the C code
    // write to global roi stack
    rois[roi_ix * npx * npy + i * npy + j] = phot * fv;
    
    // atomicAdd(&sum_array[roi_ix], phot * fv);
    return;
}

// kernel to compute psf for a single emitter
__global__
auto kernel_roi(spline *sp, float *rois, const int npx, const int npy, const float* xc_, 
const float* yc_, const float* zc_, const float* phot_) -> void {

    int r = blockIdx.x;  // roi number 'r'

    int x0, y0, z0;
    float x_delta,y_delta,z_delta;

    float xc = xc_[r];
    float yc = yc_[r];
    float zc = zc_[r];
    float phot = phot_[r];

    /* Compute delta. Will be the same for all following px */
    // floats use floorf I think, stolen from another branch
    x0 = (int)floorf(xc);
    x_delta = xc - x0;

    y0 = (int)floorf(yc);
    y_delta = yc - y0;

    z0 = (int)floorf(zc);
    z_delta = zc - z0;


    int n_threads = min(1024, npx * npy);  // max number of threads per block
    int n_blocks = ceil(static_cast<float>(npx * npy) / static_cast<float>(n_threads));

    fAt3Dj<<<n_blocks, n_threads>>>(sp, rois, r, npx, npy, x0, y0, z0, phot, x_delta, y_delta, z_delta);

    return;
}

__global__
auto kernel_derivative_roi(spline *sp, float *rois, float *drv_rois, const int npx, const int npy,
    const float *xc_, const float *yc_, const float *zc_, const float *phot_, const float *bg_, const bool add_bg) -> void {

    int r = blockIdx.x;  // roi number 'r'

    int x0, y0, z0;
    float x_delta,y_delta,z_delta;

    float xc = xc_[r];
    float yc = yc_[r];
    float zc = zc_[r];
    float phot = phot_[r];
    float bg = bg_[r];

    /* Compute delta. Will be the same for all following px */
    x0 = (int)floorf(xc);
    x_delta = xc - x0;

    y0 = (int)floorf(yc);
    y_delta = yc - y0;

    z0 = (int)floorf(zc);
    z_delta = zc - z0;

    int n_threads = min(1024, npx * npy);  // max number of threads per block
    int n_blocks = ceil(static_cast<float>(npx * npy) / static_cast<float>(n_threads));

    kernel_derivative<<<n_blocks, n_threads>>>(sp, rois, drv_rois, r, npx, npy, x0, y0, z0, phot, bg, x_delta, y_delta, z_delta, add_bg);

    return;
}

__global__
auto kernel_derivative(spline *sp, float *rois, float *drv_rois, const int roi_ix, const int npx, const int npy,
    int xc, int yc, int zc, const float phot, const float bg, const float x_delta, const float y_delta, const float z_delta, const bool add_bg) -> void {

    int i = (blockIdx.x * blockDim.x + threadIdx.x) / npx; // Number of threads I guess
    int j = (blockIdx.x * blockDim.x + threadIdx.x) % npx;

     // allocate space for df, dxf, dyf, dzf
    __shared__ float delta_f[64];
    __shared__ float dxf[64];
    __shared__ float dyf[64];
    __shared__ float dzf[64];

    float dudt[5] = { 0 };  // derivatives in this very pixel

    // term common to all pixels, must be executed at least once per kernel block (since sync only syncs within block)
    // if (i == 0 and j == 0) {  // linear / C++ equivalent
    if (threadIdx.x == 0) {

        for (int k = 0; k < 64; k++) {
            delta_f[k] = 0.0;
            dxf[k] = 0.0;
            dyf[k] = 0.0;
            dzf[k] = 0.0;
        }

        // This is different to the C library since we needed to rearrange a bit to account for the GPU parallelism
        kernel_computeDelta3D(sp, delta_f, dxf, dyf, dzf, x_delta, y_delta, z_delta);
    }
    __syncthreads();  // wait so that all threads see the deltas

    // kill excess threads
    if ((i >= npx) || (j >= npy)) {
        return;
    }

    // let each thread go to their respective pixel
    xc = xc + i;
    yc = yc + j;

    // set epsilon values outside of ROI
    if ((xc < 0) || (xc > sp->xsize-1) || (yc < 0) || (yc > sp->ysize-1)) {

        for (int k = 0; k < sp->n_par; k++) {
            dudt[k] = sp->roi_out_deriv_eps;
        }

        if (add_bg) {
            rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps + bg;
        }
        else {
            rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps;
        }
        return;
    }

    // safety for zc
    zc = max(zc,0);
    zc = min(zc,sp->zsize-1);

    float fv;  // taken from yiming, not entirely understood by myself

    // actual derivative computation
    for (int k = 0; k < 64; k++)
    {
        fv += delta_f[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[0] += dxf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[1] += dyf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[4] += dzf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
    }

    dudt[0] *= -1 * phot;
    dudt[1] *= -1 * phot;
    dudt[4] *= phot;
    dudt[2] = fv;
    dudt[3] = 1;

    // write to global roi and derivate stack
    if (add_bg) {
        rois[roi_ix * npx * npy + i * npy + j] = phot * fv + bg;
    }
    else {
        rois[roi_ix * npx * npy + i * npy + j] = phot * fv;
    }

    for (int k = 0; k < sp->n_par; k++) {
        drv_rois[roi_ix * sp->n_par * npx * npy + k * npx * npy + i * npy + j] = dudt[k];
    }

    return;
}

// accumulate rois to frames
__global__
auto roi_accumulate(float *frames, const int frame_size_x, const int frame_size_y, const int n_frames,
                    const float *rois, const int n_rois,
                    const int *frame_ix, const int *x0, const int *y0,
                    const int roi_size_x, const int roi_size_y) -> void {

        // kernel ix
        const long kx = (blockIdx.x * blockDim.x + threadIdx.x);
        if (kx >= n_rois * roi_size_x * roi_size_y) {
            return;
        }

        // roi index
        const long j = kx % roi_size_y;
        const long i = ((kx - j) / roi_size_y) % roi_size_x;
        const long r = (((kx - j) / roi_size_y) - i) / roi_size_x;

        const long ii = x0[r] + i;
        const long jj = y0[r] + j;

        if ((frame_ix[r] < 0) || (frame_ix[r] >= n_frames)) {  // if frame ix is outside
            return;
        }

        if ((ii < 0) || (jj < 0) || (ii >= frame_size_x) || (jj >= frame_size_y)) {  // if outside frame throw away
            return;
        }
        float val = rois[r * roi_size_x * roi_size_y + i * roi_size_y + j];
        
        // Possible reduction?
        atomicAdd(&frames[frame_ix[r] * frame_size_x * frame_size_y + ii * frame_size_y + jj], val);  // otherwise race condition

        return;
    }

__global__
void kernel_sum_up(spline *sp, float *rois, float* sum_array,const int npx, const int npy){
    
    int r = blockIdx.x; 

    const int np = npx * npy;
    int n_threads;
    if (np > 1024){n_threads = 1024;}
    else if(np > 512) {n_threads = 512;}
    else if(np > 256) {n_threads = 256;}
    else if(np > 128) {n_threads = 128;}
    else if(np > 64) {n_threads = 64;}
    else if(np > 32) {n_threads = 32;}
    else if(np > 16) {n_threads = 16;}
    else if(np > 8) {n_threads = 8;}
    else if(np > 4) {n_threads = 4;}

    // n_threads should be smallest power of 2 larger than half of npx*npy
    const int startPos = r*np;

    reduce_small<<<1,n_threads, 2*n_threads*sizeof(float)>>>(rois, sum_array, r, np, 1, startPos);
   

    }


__device__ void warpReduce(volatile float* sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
    }

__global__ void reduce_small(float *rois, float *g_odata, int r, const int np, int n_blocks, const int startPos) {
    extern __shared__ float sdata[];
    int tid = int(threadIdx.x);
    int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    if (i < (np)-blockDim.x) {
        sdata[tid] = rois[startPos + i] + rois[startPos + i + blockDim.x];
    } else {
        sdata[tid] = rois[startPos + i];
    }

    __syncthreads();


    // do reduction in shared mem
    for (int s=blockDim.x/2; s>32; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }       
        __syncthreads();
    }

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) {
        g_odata[r*n_blocks + blockIdx.x] = sdata[0];
        }
    }


__global__ void kernel_normalize(float *rois, float *factors, int npx, int npy){
    int r = blockIdx.x;
    float __shared__ factor; 
    if (threadIdx.x == 0){
        factor = factors[r];
    }

    int n_threads = min(1024, npx * npy);  // max number of threads per block
    int n_blocks = ceil(static_cast<float>(npx * npy) / static_cast<float>(n_threads));
    const int n_pixels = npx * npy;
    normalize<<<n_blocks, n_threads>>>(rois, factor, r, n_pixels);

}
__global__ void normalize(float *rois, float factor, int r, const int n_pixels){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int startPos = r*n_pixels;

    if (i < n_pixels) {rois[startPos + i] *= factor;}
    
}

__global__ void get_factor(const float* phot_, float *total_sum, float *factors){
    int r = blockIdx.x;
    float phot = phot_[r];
    

    factors[r] = phot / total_sum[r];

}

